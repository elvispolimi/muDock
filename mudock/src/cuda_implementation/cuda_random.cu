#include "hip/hip_runtime.h"
#include <mudock/cuda_implementation/cuda_random.cuh>

namespace mudock {
  // __device__ unsigned int hash(unsigned int x) {
  //   return (x ^ (x >> 16)) * 0x45d9f301; // A simple hash function
  // }

  __global__ void init_curand(hiprandState *state, const long seed, const int num_elements) {
    const int id     = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = gridDim.x * blockDim.x;
    for (int index = id; index < num_elements; index += stride) {
      // const int seed = hash(seed+index); // Create a unique seed for each thread
      hiprand_init(seed + index, index, 0, &state[index]);
    }
  }

  void cuda_random_object::alloc(const std::size_t num_elements) {
    const bool init = num_elements > cuda_object<hiprandState>::num_elements();
    cuda_object<hiprandState>::alloc(num_elements);
    if (init) {
      init_curand<<<4, 128>>>(cuda_object<hiprandState>::dev_pointer(),
                              std::chrono::high_resolution_clock::now().time_since_epoch().count(),
                              cuda_object<hiprandState>::num_elements());
      MUDOCK_CHECK_KERNELCALL();
      MUDOCK_CHECK(hipDeviceSynchronize());
    }
  };
} // namespace mudock
