#include "hip/hip_runtime.h"
#include <mudock/cuda_implementation/cuda_random.cuh>

namespace mudock {
  __global__ void init_curand(hiprandState *state, const int num_elements) {
    const int id     = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = gridDim.x * blockDim.x;
    for (int index = id; index < num_elements; index += stride) hiprand_init(clock64(), id, 0, &state[id]);
  }

  void cuda_random_object::alloc(const std::size_t num_elements) {
    const bool init = num_elements > cuda_object<hiprandState>::num_elements();
    cuda_object<hiprandState>::alloc(num_elements);
    if (init) {
      init_curand<<<32, 128>>>(cuda_object<hiprandState>::dev_pointer(),
                               cuda_object<hiprandState>::num_elements());
      MUDOCK_CHECK_KERNELCALL();
      MUDOCK_CHECK(hipDeviceSynchronize());
    }
  };
} // namespace mudock