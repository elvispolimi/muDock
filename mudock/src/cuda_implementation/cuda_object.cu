#include <hip/hip_runtime.h>
#include <mudock/cpp_implementation/chromosome.hpp>
#include <mudock/cuda_implementation/cuda_check_error_macro.cuh>
#include <mudock/cuda_implementation/cuda_object.cuh>
#ifdef MUDOCK_ENABLE_POLY
#include <polygeist/cuda_random.cuh>
#else
#include <mudock/cuda_implementation/cuda_random.cuh>
#endif
#include <mudock/type_alias.hpp>

namespace mudock {
  template<class T>
  cuda_object<T>::cuda_object(cuda_object&& other) {
    dev_ptr       = other.dev_ptr;
    size          = other.size;
    other.dev_ptr = nullptr;
    other.size    = 0;
  }

  // TODO destructor not supported by Polygeist 
  #ifndef MUDOCK_ENABLE_POLY
    template<class T>
    cuda_object<T>::~cuda_object() noexcept(false){
      if (dev_ptr != nullptr)
        MUDOCK_CHECK(hipFree(dev_ptr));
    }
  #endif

  template<class T>
  void cuda_object<T>::alloc(const size_t num_elements) {
    if (size < num_elements) {
      if (dev_ptr != nullptr)
        MUDOCK_CHECK(hipFree(dev_ptr));
      MUDOCK_CHECK(hipMalloc(&dev_ptr, sizeof(T) * num_elements));
    }
    size = num_elements;
  }
  template<class T>
  void cuda_object<T>::set_to_value(const int value) {
    MUDOCK_CHECK(hipMemset(dev_ptr, value, sizeof(T) * size));
  }

  template<class T>
  void cuda_object<T>::copy_host2device(const T* const host) {
    MUDOCK_CHECK(hipMemcpy(dev_ptr, host, sizeof(T) * size, hipMemcpyHostToDevice));
  }
  template<class T>
  void cuda_object<T>::copy_device2host(T* const host) const {
    MUDOCK_CHECK(hipMemcpy(host, dev_ptr, sizeof(T) * size, hipMemcpyDeviceToHost));
  }

  template<class T>
  [[nodiscard]] T* cuda_object<T>::dev_pointer() const {
    return dev_ptr;
  }
  template<class T>
  [[nodiscard]] std::size_t cuda_object<T>::num_elements() const {
    return size;
  }

  template class cuda_object<int>;
  template class cuda_object<fp_type>;
  template class cuda_object<fp_type*>;
#ifdef MUDOCK_ENABLE_POLY
  template class cuda_object<XORWOWState>;
#else
  template class cuda_object<hiprandState>;
  template class cuda_object<hipTextureObject_t>;
#endif
  template class cuda_object<chromosome>;
} // namespace mudock
